#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<math.h>
#include <hip/hip_runtime.h>

__global__ void print_kernel()
{
int i = blockIdx.x;
printf("Hello from the block ID: %d\n",i);
}

int main()
{
print_kernel<<<2,24>>>();
hipDeviceSynchronize();
return 1;
}

