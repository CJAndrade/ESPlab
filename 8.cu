#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<math.h>
#include <hip/hip_runtime.h>
#define N 256

__global__ void mul(float *da, float *db, float *dc)
{
	int row = threadIdx.y+ blockIdx.y*blockDim.y;
	int col = threadIdx.x+ blockIdx.x*blockDim.x;
	int i;
	float temp =0;
	for(i=0;i<N;i++)
	{
	temp=temp+da[row*N+i]*db[col+i*N];
	}
	__syncthreads();
	dc[row*N+col]=temp;
	printf(" dc[%d][%d]=%f \n",row,col,temp);
}

int main()
{
float *a,*b,*c;
float *da,*db,*dc;
int i,j;
int size;
size=N*N*sizeof(float);
a=(float*)malloc(N*N*sizeof(float));
b=(float*)malloc(N*N*sizeof(float));
c=(float*)malloc(N*N*sizeof(float));

	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
		a[i*N+j] = i/pow(99,i)+j*pow(0.3,j);
		b[i*N+j] =i/33+pow(3.3,j)/pow(2.2,i);
		}
	}

hipMalloc((void**)&da,size);
hipMalloc((void**)&db,size);
hipMalloc((void**)&dc,size);
hipMemcpy(da,a,size,hipMemcpyHostToDevice);
hipMemcpy(db,b,size,hipMemcpyHostToDevice);
dim3 blk(2,2,1);
dim3 thr(8,8,1);
clock_t s = clock();
mul<<<blk,thr>>>(da,db,dc);
clock_t e = clock();
hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

double time_used = (double)(e-s)/CLOCKS_PER_SEC;
printf("\nTime Spent=%f\n",time_used);

hipFree(da);
hipFree(db);
hipFree(dc);
}


