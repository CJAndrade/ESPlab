#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<math.h>
#include<malloc.h>
#include <hip/hip_runtime.h>

#define N 7000
#define TILE_WIDTH 16

//Chapter 5 page 112

__global__ void hello(float*da,float*db,float*dc)
{
 __shared__ float as[TILE_WIDTH][TILE_WIDTH];
 __shared__ float bs[TILE_WIDTH][TILE_WIDTH];
 int row=blockIdx.y*blockDim.y+threadIdx.y;
 int col=blockIdx.x*blockDim.x+threadIdx.x;
 float temp=0;
 int tx=threadIdx.x;
 int ty=threadIdx.y;

 if(row<N && col<N)
 for(int tilenum=0;tilenum<N/TILE_WIDTH;tilenum++)
 {
	as[ty][tx]=da[row*N+tilenum*TILE_WIDTH+tx];
	bs[ty][tx]=db[col+ty*N+tilenum*TILE_WIDTH*N];
	__syncthreads();
	for(int i=0;i<TILE_WIDTH;i++)
	{
		temp=temp+as[ty][i]*bs[i][tx];
	}
	__syncthreads();
	dc[row*N+col]=temp;
 }
 printf("dc[%d][%d]=%f\n",row,col,temp);

}

int main()
{
	int i,j;
	float *da,*db,*dc,*a,*b,*c;
	int size;
	size=N*N*sizeof(float);

	a=(float*)malloc(N*N*sizeof(float));
	b=(float*)malloc(N*N*sizeof(float));
	c=(float*)malloc(N*N*sizeof(float));
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			a[i*N+j]=i/pow(99,i)*pow(0.3,j);
			b[i*N+j]=i/33+pow(3.3,j)/pow(2.2,i);
		}

	}
	hipMalloc((void**)&da,size);
	hipMalloc((void**)&db,size);
	hipMalloc((void**)&dc,size);
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	dim3 blk(2,2,1);
	dim3 thr(8,8,1);
	clock_t s=clock();
	hello<<<blk,thr>>>(da,db,dc);
	clock_t e=clock();

	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
	hipFree(da);
	hipFree(db);
	hipFree(dc);

double time_used=(double)(e-s)/CLOCKS_PER_SEC;
printf("\ntime spent=%f\n",time_used);
hipDeviceSynchronize();
return 0;
}


