#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<math.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
int i=threadIdx.x;
printf("hello from %d\n",i);
}

int main()
{
hello<<<1,32>>>();
hipDeviceSynchronize();
return 0;
}
