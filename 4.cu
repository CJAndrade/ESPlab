#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<math.h>
#include <hip/hip_runtime.h>
__global__ void print_kernel()
{
printf("block= %d,thread=%d\n",blockIdx.x,threadIdx.x);
}
int main()
{
print_kernel<<<2,24>>>();
hipDeviceSynchronize();
return 1;
}

