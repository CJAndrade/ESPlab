#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<math.h>
#include <hip/hip_runtime.h>
#define N 256

__global__ void calc(float *da)
{
int m = threadIdx.x;
int temp1=0,temp2=0,temp3=0;
	if(m==0){
		temp1=da[255];
		temp2=da[254];
		temp3=da[253];
	 }
	else if(m==1){
		temp1=da[0];
		temp2=da[255];
		temp3=da[253];
	 }
	else if(m==2){
		temp1=da[1];
		temp2=da[0];
		temp3=da[255];
	 }
	else
	{
		temp1=da[(m-1)%N];
		temp2=da[(m-2)%N];
		temp3=da[(m-3)%N];
	 }
//__syncthreads();
da[m]=temp1+temp2+temp3;
printf("a[%d]=%f\n",m,da[m]);

}

int main()
{
int i;
int size;
float *a;
float *da;
size=N*sizeof(float);
a=(float*)malloc(N*sizeof(float));
for(int i=0;i<N;i++){
	a[i]=pow(1.1,i)*pow(2.1,i+1);
}
clock_t s=clock();
hipMalloc((void**)&da,size);
hipMemcpy(da,a,size,hipMemcpyHostToDevice);
calc<<<1,N>>>(da);
hipMemcpy(a,da,size,hipMemcpyDeviceToHost);
clock_t e=clock();
double time_used=(double)(e-s)/CLOCKS_PER_SEC;
printf("\n time spent =%f\n", time_used);
hipFree(da);
}


